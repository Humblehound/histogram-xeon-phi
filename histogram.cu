#include "hip/hip_runtime.h"

// System includes
#include <stdio.h>
#include "BmpImage.h"

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

using namespace std;

const int BIN_COUNT=256;
const int THREADS_PER_BLOCK=256;

int BLOCK_COUNT=8000;

void histogram_cpu(int *host_data, size_t data_size, int* histogram){
    for(int i=0;i<data_size;i++){
        histogram[host_data[i]]++;
    }
}
    
__global__ void histogram(const int *d_data, int *d_results, size_t dataSize, int threadCount, int pixelsPerThread)
{
       __shared__ unsigned int local_results[256];
       local_results[threadIdx.x] = 0;
       __syncthreads();

      int id = blockDim.x * blockIdx.x + threadIdx.x;
      int startId = pixelsPerThread*id;
      int endId = startId+pixelsPerThread;

      if(endId > dataSize){
        endId = dataSize;
      }

      for(int i=startId;i<endId;i++){
        atomicAdd(&local_results[d_data[i]], 1);        
      }
      __syncthreads();

      atomicAdd(&(d_results[threadIdx.x]), local_results[threadIdx.x] );
 }

int main(int argc, char **argv)
{

    int *host_data, *cpu_data, *host_results, *cpu_results;
    int *d_data, *d_results;
    hipEvent_t start, stop;

    BmpImage* image = new BmpImage();
    image->Load("sample.bmp");

    printf("Image size: %dx%d\n", image->GetWidth(), image->GetHeight());
    printf("Data size : %d Bytes\n", image->GetSize());
    vector<int> pixelLuminosity = image->GetLuminosityVector();;

    size_t dataSize = pixelLuminosity.size();
    int arraySize = dataSize * sizeof(int);

    host_data = (int*) malloc(arraySize);
    cpu_data = (int*) malloc(arraySize);
    host_results = (int*) calloc(BIN_COUNT, sizeof(int));
    cpu_results = (int*) calloc(BIN_COUNT, sizeof(int));

    std::copy(pixelLuminosity.begin(), pixelLuminosity.end(), host_data);
    std::copy(pixelLuminosity.begin(), pixelLuminosity.end(), cpu_data);

    histogram_cpu(cpu_data, dataSize, cpu_results);

    hipEventCreate(&start);
    hipEventCreate(&stop);

    checkCudaErrors(hipMalloc((void **)&d_results, BIN_COUNT*sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&d_data, arraySize));
    checkCudaErrors(hipMemset(d_results, 0, BIN_COUNT*sizeof(int)));
    checkCudaErrors(hipMemcpy(d_data, host_data, arraySize, hipMemcpyHostToDevice));

    int threadCount = BLOCK_COUNT * THREADS_PER_BLOCK;
    int pixelsPerThread = (dataSize/threadCount) + 1;

    printf("CUDA kernel launch with %d blocks\n", BLOCK_COUNT);
    printf("A total of %d threads launched\n", threadCount);
    printf("Pixels per thead: %d\n", pixelsPerThread);

    hipEventRecord(start);
    histogram<<<BLOCK_COUNT, THREADS_PER_BLOCK>>>(d_data, d_results, dataSize, threadCount, pixelsPerThread);
    hipEventRecord(stop);
    getLastCudaError("histogram execution failed\n");
    checkCudaErrors(hipMemcpy(host_results, d_results, BIN_COUNT*sizeof(int), hipMemcpyDeviceToHost));

    int diff = 0;

    for(int i=0;i<BIN_COUNT;i++){
        if(cpu_results[i] != host_results[i]){
            diff+=cpu_results[i]-host_results[i];
        }
    }
    if(diff != 0){
        printf("Histogram calculation error, total sum difference: %d\n", diff);
    }

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Elapsed time: %4.2fms", milliseconds);
    printf("Done\n");
    exit(0);
}